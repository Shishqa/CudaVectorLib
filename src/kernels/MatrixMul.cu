#include "hip/hip_runtime.h"
#include <kernels/MatrixMul.cuh>

__global__
void MatrixMul(size_t resultH, size_t resultW, size_t commonSide,
               hipPitchedPtr A, hipPitchedPtr B,
               hipPitchedPtr result)
{
  const size_t i = blockIdx.y * blockDim.y + threadIdx.y;
  const size_t j = blockIdx.x * blockDim.x + threadIdx.x;

  const size_t bi = threadIdx.y;
  const size_t bj = threadIdx.x;

  extern __shared__ float s[];
  float *windowA = s;
  float *windowB = s + blockDim.x * blockDim.y;

  float sum = 0;
  for (size_t k = 0; k < commonSide; k += blockDim.x) {
    float *A_row = (float *)((char *)A.ptr + i * A.pitch);
    float *B_row = (float *)((char *)B.ptr + (k + bi) * B.pitch);
    windowA[bj + bi * blockDim.x] = k + bj < commonSide && i < resultH ? A_row[k + bj] : 0;
    windowB[bi + bj * blockDim.x] = k + bi < commonSide && j < resultW ? B_row[j] : 0;
    __syncthreads();

    for (size_t bk = 0; bk < blockDim.x; ++bk) {
      sum += windowA[bk + bi * blockDim.x] * windowB[bk + bj * blockDim.x];
    }
    __syncthreads();
  }

  ((float *)((char *)result.ptr + i * result.pitch))[j] = sum;
}
