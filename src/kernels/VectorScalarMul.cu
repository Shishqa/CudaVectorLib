#include "hip/hip_runtime.h"
#include <kernels/VectorScalarMul.cuh>

__global__
void ScalarMulBlock(int numElements, float *vector1, float *vector2, float *result)
{
  extern __shared__ int s[];
  const size_t index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index < numElements) {
    s[index] = vector1[index] * vector2[index];
  }
  __syncthreads();

  for (size_t step = numElements >> 1; step > 0; step >>= 1) {
    if (index < step) {
      s[index] = s[index] + s[index + step];
    }
    __syncthreads();
  }

  if (index == 0) {
    result[blockIdx.x] = s[0];
  }
}

