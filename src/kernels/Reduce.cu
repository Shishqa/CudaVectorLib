#include "hip/hip_runtime.h"
#include <kernels/Reduce.cuh>


__global__
void AccumulateToBlock(size_t numElements, float *vector, float *result)
{
  const size_t index = blockDim.x * blockIdx.x + threadIdx.x;
  const size_t stride = blockDim.x * gridDim.x;

  float sum = 0;
  for (size_t i = index; i < numElements; i += stride) {
    sum += vector[i];
  }

  if (index < numElements) result[index] = sum;
}

__global__
void ReduceOnBlock(int numElements, float *arr, float *result)
{
  extern __shared__ int s[];
  const size_t tid = threadIdx.x;
  const size_t i = blockDim.x * blockIdx.x * 2 + threadIdx.x;

  if (i + blockDim.x < numElements) {
    s[tid] = arr[i] + arr[i + blockDim.x];
  } else if (i < numElements) {
    s[tid] = arr[i];
  } else {
    s[tid] = 0;
  }
  __syncthreads();

  for (size_t step = blockDim.x >> 1; step > 0; step >>= 1) {
    if (tid < step) {
      s[tid] += s[tid + step];
    }
    __syncthreads();
  }

  if (tid == 0) {
    result[blockIdx.x] = s[0];
  }
}
