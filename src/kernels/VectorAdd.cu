#include "hip/hip_runtime.h"
#include <kernels/VectorAdd.cuh>

__global__
void KernelAdd(size_t numElements, float *x, float *y, float *result)
{
  const size_t index = blockIdx.x * blockDim.x + threadIdx.x;
  const size_t stride = blockDim.x * gridDim.x;

  for (size_t i = index; i < numElements; i += stride) {
    result[i] = x[i] + y[i];
  }
}
