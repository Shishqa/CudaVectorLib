#include "hip/hip_runtime.h"
#include <kernels/MatrixAdd.cuh>

__global__
void KernelMatrixAdd(size_t height, size_t width, hipPitchedPtr A, hipPitchedPtr B, hipPitchedPtr result)
{
  const size_t y = blockIdx.y * blockDim.y + threadIdx.y;
  const size_t x = blockIdx.x * blockDim.x + threadIdx.x;

  const size_t stride_y = gridDim.y * blockDim.y;
  const size_t stride_x = gridDim.x * blockDim.x;

  for (size_t i = y; i < height; i += stride_y) {
    const float *A_row = (float *)((char *)A.ptr + i * A.pitch);
    const float *B_row = (float *)((char *)B.ptr + i * B.pitch);
    float *R_row = (float *)((char *)result.ptr + i * result.pitch);
    for (size_t j = x; j < width; j += stride_x) {
      R_row[j] = A_row[j] + B_row[j];
    }
  }
}

