#include "hip/hip_runtime.h"
#include <kernels/VectorMul.cuh>

__global__
void KernelMul(size_t numElements, float *x, float *y, float *result)
{
  const size_t index = blockIdx.x * blockDim.x + threadIdx.x;
  const size_t stride = blockDim.x * gridDim.x;

  for (size_t i = index; i < numElements; i += stride) {
    result[i] = x[i] * y[i];
  }
}

