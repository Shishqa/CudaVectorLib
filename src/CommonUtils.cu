#include <cstdio>

#include <CommonUtils.cuh>

float MeasureTime(Routine routine)
{
  hipEvent_t start;
  hipEvent_t stop;

  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  routine();
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  float elapsed = 0;
  hipEventElapsedTime(&elapsed, start, stop);

  return elapsed;
}

void PrintMatrix(float *m, dim3 size, const char *sep)
{
  for (size_t i = 0; i < size.y; ++i) {
    for (size_t j = 0; j < size.x; ++j) {
      printf("%f%s", m[j + size.x * i], sep);
    }
    printf("\n");
  }
  printf("\n");
}

void FillMatrix(float *m, dim3 size, float value)
{
  for (size_t i = 0; i < size.x * size.y; ++i) {
    m[i] = value;
  }
}

void Report(size_t array_size, size_t block_size, float time)
{
  printf("%lu,%lu,%f\n", array_size, block_size, time);
}
