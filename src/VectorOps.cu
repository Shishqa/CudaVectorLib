#include "hip/hip_runtime.h"
#include <cassert>

#include <VectorOps.cuh>

#include <kernels/VectorAdd.cuh>
#include <kernels/VectorMul.cuh>
#include <Reduce.cuh>

#include <CommonUtils.cuh>


float VectorAdd(size_t block_size, const Vector& a, const Vector& b,
                Vector& res) {
  assert(a.size_ == b.size_);
  const size_t grid_size = (a.size_ + block_size - 1) / block_size;

  return MeasureTime([&]() {
    KernelAdd<<<grid_size, block_size>>>(a.size_, a.device_data_, b.device_data_, res.device_data_);
  });
}


float VectorMul(size_t block_size, const Vector& a, const Vector& b,
                Vector& res) {
  assert(a.size_ == b.size_);
  const size_t grid_size = (a.size_ + block_size - 1) / block_size;

  return MeasureTime([&]() {
    KernelMul<<<grid_size, block_size>>>(a.size_, a.device_data_, b.device_data_, res.device_data_);
  });
}


float VectorDot(size_t block_size, const Vector& a, const Vector& b,
                float& res, Reducer reducer) {
  assert(a.size_ == b.size_);
  Vector tmp(a.size_);

  size_t grid_size = (a.size_ + block_size - 1) / block_size;
  float elapsed = MeasureTime([&]() {
    KernelMul<<<grid_size, block_size>>>(a.size_, a.device_data_, b.device_data_, tmp.device_data_);
    reducer(block_size, tmp);
  });

  tmp.Fetch(1, &res);
  return elapsed;
}


float VectorCos(size_t block_size, const Vector& a, const Vector& b,
                float &res, Reducer reducer)
{
  float a_dot_b, a_dot_a, b_dot_b;
  float elapsed_1 = VectorDot(block_size, a, b, a_dot_b, reducer);
  float elapsed_2 = VectorDot(block_size, a, a, a_dot_a, reducer);
  float elapsed_3 = VectorDot(block_size, b, b, b_dot_b, reducer);

  res = a_dot_b / sqrt(a_dot_a * b_dot_b);
  return elapsed_1 + elapsed_2 + elapsed_3;
}

