#include <Matrix.cuh>


Matrix::Matrix(dim3 size)
    : size_(size) {
  hipExtent extent = make_hipExtent(size.x * sizeof(float), size.y, 1);
  hipMalloc3D(&device_data_, extent);
}


Matrix::Matrix(dim3 size, const float *host_data)
    : size_(size) {
  hipExtent extent = make_hipExtent(size.x * sizeof(float), size.y, 1);
  hipMalloc3D(&device_data_, extent);
  hipMemcpy2D(device_data_.ptr, device_data_.pitch, host_data, size.x * sizeof(*host_data), size.x * sizeof(*host_data), size.y, hipMemcpyHostToDevice);
}


Matrix::~Matrix() {
  if (device_data_.ptr) {
    hipFree(device_data_.ptr);
  }
}


Matrix::Matrix(Matrix&& other)
    : size_(other.size_), device_data_(other.device_data_) {
  other.device_data_.ptr = nullptr;
  other.size_ = dim3(0, 0, 0);
}


Matrix& Matrix::operator=(Matrix&& other) {
  if (&other == this) {
    return *this;
  }

  device_data_ = other.device_data_;
  size_ = other.size_;

  other.device_data_.ptr = nullptr;
  other.size_ = dim3(0, 0, 0);

  return *this;
}


dim3 Matrix::Fetch(dim3 size, float *host_data) {
  hipMemcpy2D(host_data, size.x * sizeof(*host_data), device_data_.ptr, device_data_.pitch, size.x * sizeof(*host_data), size.y, hipMemcpyDeviceToHost);
  return size;
}


