#include <cassert>

#include <Vector.cuh>

#include <kernels/VectorAdd.cuh>
#include <kernels/VectorMul.cuh>
#include <Reduce.cuh>

#include <CommonUtils.cuh>


Vector::Vector(size_t size)
    : size_(size) {
  hipMalloc(&device_data_, size * sizeof(*device_data_));
}


Vector::Vector(size_t size, const float *host_data)
    : size_(size) {
  hipMalloc(&device_data_, size * sizeof(*device_data_));
  hipMemcpy(device_data_, host_data, size * sizeof(*host_data),
             hipMemcpyHostToDevice);
}


Vector::~Vector() {
  if (device_data_) {
    hipFree(device_data_);
  }
}


Vector::Vector(Vector&& other)
    : size_(other.size_), device_data_(other.device_data_) {
  other.device_data_ = nullptr;
  other.size_ = 0;
}


Vector& Vector::operator=(Vector&& other) {
  if (&other == this) {
    return *this;
  }

  device_data_ = other.device_data_;
  size_ = other.size_;

  other.device_data_ = nullptr;
  other.size_ = 0;

  return *this;
}


size_t Vector::Fetch(size_t size, float *host_data) {
  size_t fetch_size = (size > size_) ? size_ : size;
  hipMemcpy(host_data, device_data_, fetch_size * sizeof(*host_data), hipMemcpyDeviceToHost);
  return fetch_size;
}


