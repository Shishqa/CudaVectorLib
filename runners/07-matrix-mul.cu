#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <cassert>

#include <MatrixOps.cuh>
#include <CommonUtils.cuh>

void PrintUsage(const char *programName)
{
  printf("usage: %s matrix_size block_size\n\n", programName);
}

int main(int argc, char *argv[])
{
  if (argc != 3) {
    PrintUsage(argc == 0 ? "matrix_mul" : argv[0]);
    return 0;
  }

  size_t matrixSz = strtoull(argv[1], NULL, 10);
  dim3 size(matrixSz, matrixSz);

  size_t blockSz = strtoull(argv[2], NULL, 10);
  dim3 blockSize(blockSz, blockSz);

  float *r = new float[size.x * size.y];
  float *x = new float[size.x * size.y];
  float *y = new float[size.x * size.y];

  static const float X_VAL = 1.0f;
  static const float Y_VAL = 2.0f;
  FillMatrix(x, size, X_VAL);
  FillMatrix(y, size, Y_VAL);

  Matrix a(size, x);
  Matrix b(size, y);

  delete[] x;
  delete[] y;

  Matrix c(size);
  float elapsed = MatrixMul(blockSize, a, b, c);
  Report(matrixSz, blockSz, elapsed);

  c.Fetch(size, r);

  float maxError = 0.0f;
  for (size_t i = 0; i < size.y; ++i) {
    for (size_t j = 0; j < size.x; ++j) {
      maxError = fmax(maxError, fabs(r[size.x * i + j] - size.x * Y_VAL));
    }
  }
  assert(maxError < 0.001);

  delete[] r;
  return 0;
}
