#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <cassert>

#include <MatrixOps.cuh>
#include <CommonUtils.cuh>

void PrintUsage(const char *programName)
{
  printf("usage: %s matrix_size block_size\n\n", programName);
}

int main(int argc, char *argv[])
{
  if (argc != 3) {
    PrintUsage(argc == 0 ? "kernel_add" : argv[0]);
    return 0;
  }

  size_t matrixSz = strtoull(argv[1], NULL, 10);
  dim3 size(matrixSz, matrixSz);

  size_t blockSz = strtoull(argv[2], NULL, 10);
  dim3 blockSize(blockSz, blockSz);

  float *x = new float[size.x * size.y];
  float *y = new float[size.x * size.y];

  static const float X_VAL = 1.0f;
  static const float Y_VAL = 2.0f;
  FillMatrix(x, size, X_VAL);
  FillMatrix(y, size, Y_VAL);

  Matrix a(size, x);
  Matrix b(size, y);

  delete[] x;
  delete[] y;

  Matrix c(size);
  float elapsed = MatrixAdd(blockSz, a, b, c);
  Report(matrixSz, blockSz, elapsed);

  float *r = new float[size.x * size.y];
  c.Fetch(size, r);

  float maxError = 0.0f;
  for (size_t i = 0; i < size.x * size.y; i++) {
    maxError = fmax(maxError, fabs(r[i] - (X_VAL + Y_VAL)));
  }
  assert(maxError < 0.001);

  delete[] r;
  return 0;
}
