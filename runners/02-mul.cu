#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <cassert>

#include <CommonUtils.cuh>
#include <VectorOps.cuh>

void PrintUsage(const char *programName)
{
  printf("usage: %s array_size block_size\n\n", programName);
}

int main(int argc, char *argv[])
{
  if (argc != 3) {
    PrintUsage(argc == 0 ? "kernel_mul" : argv[0]);
    return 0;
  }

  const size_t arrSize = strtoull(argv[1], NULL, 10);
  const size_t blockSize = strtoull(argv[2], NULL, 10);

  float *x = new float[arrSize];
  float *y = new float[arrSize];

  static const float X_VAL = 5.0f;
  static const float Y_VAL = 2.0f;
  FillMatrix(x, dim3(arrSize, 1, 1), X_VAL);
  FillMatrix(y, dim3(arrSize, 1, 1), Y_VAL);

  Vector a(arrSize, x);
  Vector b(arrSize, y);
  Vector c(arrSize);

  delete[] x;
  delete[] y;

  float elapsed = VectorMul(blockSize, a, b, c);
  Report(arrSize, blockSize, elapsed);

  float *r = new float[arrSize];
  c.Fetch(arrSize, r);

  float maxError = 0.0f;
  for (size_t i = 0; i < arrSize; i++) {
    maxError = fmax(maxError, fabs(r[i] - (X_VAL * Y_VAL)));
  }
  assert(maxError < 0.001);

  return 0;
}
